#include "hip/hip_runtime.h"
extern "C"
{
    #include "cuda_kernels.h"

// Custom CUDA GEMM kernel.
__global__ void cuda_matmul_kernel(const unsigned int M, const unsigned int N, const unsigned int K,
    const float *A, const unsigned int lda, const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    const float *Bias, LAYER_ACT activation_type)
{
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    const int id = threadIdx.x*(_BLOCK_N_SIZE / _THREAD_N_SIZE) + threadIdx.y;
    __shared__ float ACache [_BLOCK_K_SIZE*_BLOCK_M_SIZE];
    __shared__ float BCache [_BLOCK_K_SIZE*_BLOCK_N_SIZE];
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = Bias[mGroup + mLocal + vecM];
        }   
    }

    // for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    // {
    //     for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
    //     {
    //         const int m = mGroup + mLocal + vecM;
    //         const int n = nGroup + nLocal + vecN;
    //         if (m < M &&  n < N)
    //         {
    //             C[n * ldc + m] = 0;
    //             for (int k = 0; k < K; k++)
    //             {
    //                 C[n * ldc + m] += A[((m/_A_MIN_DIM)*lda + k) * _A_MIN_DIM + m%_A_MIN_DIM] * B[n * ldb + k];
    //             }
    //         }
    //     }
    // }
    int kIdx = 0;  
    if (K%_BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = A[((m/_A_MIN_DIM)*lda + k) * _A_MIN_DIM + m%_A_MIN_DIM];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B[ldb*n + k];
        }
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
        for (; kIdx < K%_BLOCK_K_SIZE; kIdx++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    // printf ("B%dT%d: (%d, %d) %3.3f, %3.3f\n", blockIdx.x + blockIdx.y
                    //     ,id, vecN, vecM, ACache[kk*_BLOCK_M_SIZE + mLocal + vecM], BCache[kk*_BLOCK_N_SIZE + nLocal + vecN]);
                    cout[vecN][vecM] += ACache[kIdx*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kIdx*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
    }
    for (; kIdx < K; kIdx += _BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = A[((m/_A_MIN_DIM)*lda + k) * _A_MIN_DIM + m%_A_MIN_DIM];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B[ldb*n + k];
        }
        __syncthreads();
        for (int kk = 0; kk < _BLOCK_K_SIZE; kk++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    cout[vecN][vecM] += ACache[kk*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kk*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
    }
    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                if (activation_type == RELU)
                    cout[vecN][vecM] = cout[vecN][vecM] > 0 ? cout[vecN][vecM] : 0;
                else if (activation_type == GELU)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + erff ((cout[vecN][vecM])*0.7071067811865475f));
                else if (activation_type == GELU_ACCURATE)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + tanhf (0.7978845608028654f * (cout[vecN][vecM] + 0.044715f * powf (cout[vecN][vecM], 3))));
                C[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
}
__global__ void cuda_preset_conv2d_ptrs_kernel(
    const unsigned int N, const unsigned int Range, float *null_data,
    int *col_idx_arr, float **col_ptr_arr, const unsigned int col_per_n, const unsigned int K_col,
    float *B, const unsigned int ldb)
{
    const int id = blockIdx.x * _BLOCK_RESIDUAL_SIZE + threadIdx.x;
    if (id >= Range)
        return;
    for (int col = 0; col < col_per_n; col++)
    {
        if (id < N && col_idx_arr[id*col_per_n + col] != -1)
        {
            col_ptr_arr[id*col_per_n + col] = B + col_idx_arr[id*col_per_n + col] * ldb;
        }   
        else 
            col_ptr_arr[id*col_per_n + col] = null_data;
    }
}
__global__ void cuda_conv2d_kernel(
    const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n, const unsigned int K_col,
    const float *A, const unsigned int lda, const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    const float *Bias, LAYER_ACT activation_type)
{
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    const int id = threadIdx.x*(_BLOCK_N_SIZE / _THREAD_N_SIZE) + threadIdx.y;
    __shared__ float ACache [_BLOCK_K_SIZE*_BLOCK_M_SIZE];
    __shared__ float BCache [_BLOCK_K_SIZE*_BLOCK_N_SIZE];
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = Bias[mGroup + mLocal + vecM];
        }   
    }

    // for (int col = 0; col < col_per_n; col++)
    // {
    //     for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    //     {
    //         for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
    //         {
    //             const int m = mGroup + mLocal + vecM;
    //             const int n = nGroup + nLocal + vecN;
    //             if (m < M &&  n < N)
    //             {
    //                 const float *B_col = B + col_idx_arr[n*col_per_n + col] * ldb;
    //                 if (col_idx_arr[n*col_per_n + col] == -1)
    //                     continue;
    //                 for (int k = 0; k < K_col; k++)
    //                 {
    //                     int A_k = col*K_col + k;
    //                     cout[vecN][vecM] += A[((m/_THREAD_M_SIZE)*lda + A_k) * _THREAD_M_SIZE + m%_THREAD_M_SIZE] * B_col[k];
    //                 }
    //             }
    //         }
    //     }
    // }

    for (int col = 0; col < col_per_n; col++)
    {
        int kIdx = 0;  
        if (K_col%_BLOCK_K_SIZE)
        {
            // Load caches.
            for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
            {
                const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
                const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
                const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
                if (m < M)
                    ACache[cache_idx] = A[((m/_THREAD_M_SIZE)*lda + col*K_col + k) * _THREAD_M_SIZE + m%_THREAD_M_SIZE];

            }
            for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
            {
                const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
                const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
                const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
                if (col_idx_arr[n*col_per_n + col] != -1)
                {
                    const float *B_col = B + col_idx_arr[n*col_per_n + col] * ldb;
                    BCache[cache_idx] = B_col[k];
                }   
                else 
                    BCache[cache_idx] = 0;
            }
            __syncthreads();
            // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
            for (; kIdx < K_col%_BLOCK_K_SIZE; kIdx++)
            {
                // Calculate.
                for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
                {
                    for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                    {
                        // printf ("B%dT%d: (%d, %d) %3.3f, %3.3f\n", blockIdx.x + blockIdx.y
                        //     ,id, vecN, vecM, ACache[kk*_BLOCK_M_SIZE + mLocal + vecM], BCache[kk*_BLOCK_N_SIZE + nLocal + vecN]);
                        cout[vecN][vecM] += ACache[kIdx*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kIdx*_BLOCK_N_SIZE + nLocal + vecN];
                    }   
                }
            }
            // Sync threads.
            __syncthreads();
            // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
        }
        for (; kIdx < K_col; kIdx += _BLOCK_K_SIZE)
        {
            // Load caches.
            for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
            {
                const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
                const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
                const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
                if (m < M)
                    ACache[cache_idx] = A[((m/_THREAD_M_SIZE)*lda + col*K_col + k) * _THREAD_M_SIZE + m%_THREAD_M_SIZE];
  
            }
            for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
            {
                const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
                const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
                const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
                if (col_idx_arr[n*col_per_n + col] != -1)
                {
                    const float *B_col = B + col_idx_arr[n*col_per_n + col] * ldb;
                    BCache[cache_idx] = B_col[k];
                }   
                else 
                    BCache[cache_idx] = 0;
            }
            __syncthreads();
            for (int kk = 0; kk < _BLOCK_K_SIZE; kk++)
            {
                // Calculate.
                for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
                {
                    for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                    {
                        cout[vecN][vecM] += ACache[kk*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kk*_BLOCK_N_SIZE + nLocal + vecN];
                    }   
                }
            }
            // Sync threads.
            __syncthreads();
        }
    }

    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                if (activation_type == RELU)
                    cout[vecN][vecM] = cout[vecN][vecM] > 0 ? cout[vecN][vecM] : 0;
                else if (activation_type == GELU)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + erff ((cout[vecN][vecM])*0.7071067811865475f));
                else if (activation_type == GELU_ACCURATE)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + tanhf (0.7978845608028654f * (cout[vecN][vecM] + 0.044715f * powf (cout[vecN][vecM], 3))));
                C[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
}
__global__ void cuda_maxpool_kernel(
    const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n,
    const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    LAYER_ACT activation_type)
{
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = -INFINITY;
        }   
    }

    for (int col = 0; col < col_per_n; col++)
    {
        for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
        {
            const int n = nGroup + nLocal + vecN;
            const float *B_col = B + col_idx_arr[n*col_per_n + col] * ldb;
            if (col_idx_arr[n*col_per_n + col] == -1)
                continue;
            for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
            {
                const int m = mGroup + mLocal + vecM;
                if (m < M &&  n < N)
                {
                    cout[vecN][vecM] = cout[vecN][vecM] > B_col[m] ? cout[vecN][vecM] : B_col[m];
                }
            }
        }
    }
    
    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                if (activation_type == RELU)
                    cout[vecN][vecM] = cout[vecN][vecM] > 0 ? cout[vecN][vecM] : 0;
                else if (activation_type == GELU)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + erff ((cout[vecN][vecM])*0.7071067811865475f));
                else if (activation_type == GELU_ACCURATE)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + tanhf (0.7978845608028654f * (cout[vecN][vecM] + 0.044715f * powf (cout[vecN][vecM], 3))));
                C[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
}
__global__ void cuda_avgpool_kernel(
    const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n,
    const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    LAYER_ACT activation_type)
{
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = 0;
        }   
    }

    for (int col = 0; col < col_per_n; col++)
    {
        for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
        {
            const int n = nGroup + nLocal + vecN;
            const float *B_col = B + col_idx_arr[n*col_per_n + col] * ldb;
            if (col_idx_arr[n*col_per_n + col] == -1)
                continue;
            for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
            {
                const int m = mGroup + mLocal + vecM;
                if (m < M &&  n < N)
                {
                    cout[vecN][vecM] += B_col[m];
                }
            }
        }
    }
    
    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                cout[vecN][vecM] = cout[vecN][vecM] / col_per_n;
                if (activation_type == RELU)
                    cout[vecN][vecM] = cout[vecN][vecM] > 0 ? cout[vecN][vecM] : 0;
                else if (activation_type == GELU)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + erff ((cout[vecN][vecM])*0.7071067811865475f));
                else if (activation_type == GELU_ACCURATE)
                    cout[vecN][vecM] = cout[vecN][vecM] * 0.5 * (1 + tanhf (0.7978845608028654f * (cout[vecN][vecM] + 0.044715f * powf (cout[vecN][vecM], 3))));
                C[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
}
__global__ void cuda_k_attention_kernel(const unsigned int num_heads, const unsigned int num_hidden, const unsigned int num_seq,
    const unsigned int M, const unsigned int N, const unsigned int K,
    const float *key, const unsigned int ldk, const float *B, const unsigned int ldb, float *C, const unsigned int ldc)
{
    const int batch = blockIdx.z/num_heads;
    const int head = blockIdx.z%num_heads;
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    const int id = threadIdx.x*(_BLOCK_N_SIZE / _THREAD_N_SIZE) + threadIdx.y;
    __shared__ float ACache [_BLOCK_K_SIZE*_BLOCK_M_SIZE];
    __shared__ float BCache [_BLOCK_K_SIZE*_BLOCK_N_SIZE];
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = 0;
        }   
    }

    const float *key_head = key + batch * ldk * M + head * K;
    const float *B_head = B + batch * ldb * N + head * K;
    float *C_head = C + batch * num_heads  * ldc * N + head * ldc * N;

    int kIdx = 0;  
    if (K%_BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = key_head[m * ldk + k];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B_head[ldb*n + k];
        }
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
        for (; kIdx < K%_BLOCK_K_SIZE; kIdx++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    // printf ("B%dT%d: (%d, %d) %3.3f, %3.3f\n", blockIdx.x + blockIdx.y
                    //     ,id, vecN, vecM, ACache[kk*_BLOCK_M_SIZE + mLocal + vecM], BCache[kk*_BLOCK_N_SIZE + nLocal + vecN]);
                    cout[vecN][vecM] += ACache[kIdx*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kIdx*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
    }
    for (; kIdx < K; kIdx += _BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = key_head[m * ldk + k];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B_head[ldb*n + k];
        }
        __syncthreads();
        for (int kk = 0; kk < _BLOCK_K_SIZE; kk++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    cout[vecN][vecM] += ACache[kk*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kk*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
    }
    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                C_head[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
    __syncthreads();

}
__global__ void cuda_k_attention_prob_kernel(const unsigned int num_heads, const unsigned int num_hidden, const unsigned int num_seq,
    const unsigned int M, const unsigned int N, const unsigned int K,
    const float *key, const unsigned int ldk, const float *B, const unsigned int ldb, float *C, const unsigned int ldc)
{
    const int batch = blockIdx.x;
    const int head = blockIdx.y;
    const int n = threadIdx.x;
    float *C_head = C + batch * num_heads  * ldc * N + head * ldc * N + n*ldc;
    float total = 0;
    for (unsigned int m = 0; m < M; m++)
    {
        C_head[m] /= sqrtf (K);
        C_head[m] = expf (C_head[m]);
        total += C_head[m];
    }
    for (unsigned int m = 0; m < M; m++)
    {
        C_head[m] /= total;
    }
}
__global__ void cuda_v_attention_kernel(const unsigned int num_heads, const unsigned int num_hidden, const unsigned int num_seq,
    const unsigned int M, const unsigned int N, const unsigned int K,
    const float *value, const unsigned int ldv, const float *B, const unsigned int ldb, float *C, const unsigned int ldc)
{
    const int batch = blockIdx.z/num_heads;
    const int head = blockIdx.z%num_heads;
    const int mLocal = threadIdx.x*_THREAD_M_SIZE;
    const int nLocal = threadIdx.y*_THREAD_N_SIZE; 
    const int mGroup = blockIdx.x*_BLOCK_M_SIZE;
    const int nGroup = blockIdx.y*_BLOCK_N_SIZE;
    const int id = threadIdx.x*(_BLOCK_N_SIZE / _THREAD_N_SIZE) + threadIdx.y;
    __shared__ float ACache [_BLOCK_K_SIZE*_BLOCK_M_SIZE];
    __shared__ float BCache [_BLOCK_K_SIZE*_BLOCK_N_SIZE];
    float cout[_THREAD_N_SIZE][_THREAD_M_SIZE];
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            cout[vecN][vecM] = 0;
        }   
    }

    const float *val_head = value + batch * ldv * K + head * M;
    const float *B_head = B + batch * num_heads * ldb * N + head * ldb * N;
    float *C_head = C + batch * ldc * N + head * M;

    int kIdx = 0;  
    if (K%_BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = val_head[k * ldv + m];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B_head[ldb*n + k];
        }
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
        for (; kIdx < K%_BLOCK_K_SIZE; kIdx++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    // printf ("B%dT%d: (%d, %d) %3.3f, %3.3f\n", blockIdx.x + blockIdx.y
                    //     ,id, vecN, vecM, ACache[kk*_BLOCK_M_SIZE + mLocal + vecM], BCache[kk*_BLOCK_N_SIZE + nLocal + vecN]);
                    cout[vecN][vecM] += ACache[kIdx*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kIdx*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
        // printf ("Thread %d: %3.3f\n", id, cout[0][0]);
    }
    for (; kIdx < K; kIdx += _BLOCK_K_SIZE)
    {
        // Load caches.
        for (int aIdx = 0; aIdx < (_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD); aIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_A_K_PER_LOAD) + aIdx;
            const int m = mGroup + cache_idx%_BLOCK_M_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_M_SIZE;
            ACache[cache_idx] = val_head[k * ldv + m];
        }
        for (int bIdx = 0; bIdx < (_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD); bIdx++)
        {
            const int cache_idx = id*(_BLOCK_K_SIZE/_CACHE_B_K_PER_LOAD) + bIdx;
            const int n = nGroup + cache_idx%_BLOCK_N_SIZE;
            const int k = kIdx + cache_idx/_BLOCK_N_SIZE;
            BCache[cache_idx] = B_head[ldb*n + k];
        }
        __syncthreads();
        for (int kk = 0; kk < _BLOCK_K_SIZE; kk++)
        {
            // Calculate.
            for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
            {
                for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
                {
                    cout[vecN][vecM] += ACache[kk*_BLOCK_M_SIZE + mLocal + vecM] * BCache[kk*_BLOCK_N_SIZE + nLocal + vecN];
                }   
            }
        }
        // Sync threads.
        __syncthreads();
    }
    // Save results
    for (int vecN = 0; vecN < _THREAD_N_SIZE; vecN++)
    {
        for (int vecM = 0; vecM < _THREAD_M_SIZE; vecM++)
        {
            const int m = mGroup + mLocal + vecM;
            const int n = nGroup + nLocal + vecN;
            if (m < M &&  n < N)
            {
                C_head[ldc*n + m] = cout[vecN][vecM];
            }
        }   
    }
    __syncthreads();
}
__global__ void cuda_residual_kernel (const unsigned int num_elements, const float *A, const float *B, float *C, LAYER_ACT activation_type)
{
    const int id = blockIdx.x * _BLOCK_RESIDUAL_SIZE + threadIdx.x;
    if (id < num_elements)
    {
        float val = A[id] + B[id];
        if (activation_type == RELU)
            val = val > 0 ? val : 0;
        else if (activation_type == GELU)
            val = val * 0.5 * (1 + erff (val*0.7071067811865475f));
        else if (activation_type == GELU_ACCURATE)
            val = val * 0.5 * (1 + tanhf (0.7978845608028654f * (val + 0.044715f * powf (val, 3))));
        C[id] = val;
    }
}
__global__ void cuda_layernorm_kernel(const float *input, const float *weight, const float *bias, 
    float *output, unsigned int N, unsigned int M, unsigned int ldb, unsigned int ldc)
{
    const int n = blockIdx.x * _BLOCK_RESIDUAL_SIZE + threadIdx.x;
    if (n >= N)
        return;
    float mean = 0;
    float var = 0;
    for (unsigned int m = 0; m < M; m++)
    {
        mean += input[n * ldb + m];
        var += input[n * ldb + m] * input[n * ldb + m];
    }
    mean /= M;
    var /= M;
    var -= mean * mean;
    var = 1 / sqrtf (var + 1e-12);
    for (unsigned int m = 0; m < M; m++)
    {
        output[n * ldc + m] = (input[n * ldb + m] - mean) * var * weight[m] + bias[m];
    }
}
void cuda_preset_conv2d_ptrs(
    const unsigned int N, const unsigned int Range, float *null_data,
    int *col_idx_arr, float **col_ptr_arr, const unsigned int col_per_n, const unsigned int K_col,
    float *B, const unsigned int ldb, hipStream_t stream)
{
    dim3 gridDim (Range/_BLOCK_RESIDUAL_SIZE + ((Range%_BLOCK_RESIDUAL_SIZE) > 0), 1, 1);
    dim3 blockDim (_BLOCK_RESIDUAL_SIZE, 1, 1);
    cuda_preset_conv2d_ptrs_kernel<<<gridDim, blockDim, 0, stream>>>(N, Range, null_data, 
    col_idx_arr, col_ptr_arr, col_per_n, K_col, B, ldb);
}
void cuda_conv2d (const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n, const unsigned int K_col,
    const float *A, const unsigned int lda, const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    const float *Bias, LAYER_ACT activation_type, hipStream_t stream)
{
    // int p1 = (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM;
    // int p2 = (_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM;
    // if (!(p1 == 0 && p2 == 0))
    // {
    //     printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
    //     p1, p2); 
    //     exit(0);
    // }

        dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), 1);
        dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
        cuda_conv2d_kernel<<<gridDim, blockDim, 0, stream>>>(M, N, col_idx_arr, col_per_n, K_col,
            A, lda, B, ldb, C, ldc, Bias, activation_type);
}
void cuda_maxpool(
    const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n,
    const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    LAYER_ACT activation_type, hipStream_t stream)
{
    #ifdef DEBUG
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
        (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM), ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM)); 
        exit(0);
    }
    #endif
        dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), 1);
        dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
        cuda_maxpool_kernel<<<gridDim, blockDim, 0, stream>>>(M, N, 
        col_idx_arr, col_per_n, B, ldb, C, ldc, activation_type);
}
void cuda_avgpool(
    const unsigned int M, const unsigned int N, 
    const int *col_idx_arr, const unsigned int col_per_n,
    const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
    LAYER_ACT activation_type, hipStream_t stream)
{
    #ifdef DEBUG
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
        (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM), ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM)); 
        exit(0);
    }
    #endif
        dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), 1);
        dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
        cuda_avgpool_kernel<<<gridDim, blockDim, 0, stream>>>(M, N, 
        col_idx_arr, col_per_n, B, ldb, C, ldc, activation_type);
}
void cuda_matmul (const unsigned int M, const unsigned int N, const unsigned int K,
		 const float *A, const unsigned int lda, const float *B, const unsigned int ldb, float *C, const unsigned int ldc,
         const float *Bias, LAYER_ACT activation_type, hipStream_t stream)
{
    #ifdef DEBUG
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
        (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM), ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM)); 
        exit(0);
    }
    #endif
        dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), 1);
        dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
        cuda_matmul_kernel<<<gridDim, blockDim, 0, stream>>>(M, N, K, A, lda, B, ldb, C, ldc, Bias, activation_type);
}
void cuda_k_attention (const float *input_1, const float *input_2, float *output, unsigned int batch_size
    , unsigned int num_heads, unsigned int num_hidden, unsigned int num_seq, hipStream_t stream)
{
    #ifdef DEBUG
    if (input_1 == NULL)
        FPRT (stderr, "Error in naive_k_attention: input_1 is NULL.\n");
    if (input_2 == NULL)
        FPRT (stderr, "Error in naive_k_attention: input_2 is NULL.\n");
    if (output == NULL)
        FPRT (stderr, "Error in naive_k_attention: output is NULL.\n");
    #endif
    const unsigned int hidden_per_head = num_hidden / num_heads;
    const unsigned int M = num_seq;
    const unsigned int N = num_seq;
    const unsigned int K = hidden_per_head;
    const unsigned int ldk = num_hidden;
    const unsigned int ldb = num_hidden;
    const unsigned int ldc = num_seq;

    #ifdef DEBUG
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
        (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM), ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM)); 
        exit(0);
    }
    #endif

    dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), num_heads*batch_size);
    dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
    cuda_k_attention_kernel<<<gridDim, blockDim, 0, stream>>> (num_heads, num_hidden, num_seq,
        M, N, K, input_2, ldk, input_1, ldb, output, ldc);
    dim3 prob_gridDim (batch_size, num_heads, 1);
    dim3 prob_blockDim (N, 1, 1);
    cuda_k_attention_prob_kernel<<<prob_gridDim, prob_blockDim, 0, stream>>> (num_heads, num_hidden, num_seq,
        M, N, K, input_2, ldk, input_1, ldb, output, ldc);
    
}
void cuda_v_attention (const float *input_1, const float *input_2, float *output, unsigned int batch_size
    , unsigned int num_heads, unsigned int num_hidden, unsigned int num_seq, hipStream_t stream)
{
    const unsigned int hidden_per_head = num_hidden / num_heads;
    const unsigned int M = hidden_per_head;
    const unsigned int N = num_seq;
    const unsigned int K = num_seq;
    const unsigned int ldv = num_hidden;
    const unsigned int ldb = num_seq;
    const unsigned int ldc = num_hidden;

    #ifdef DEBUG
    if (!(((_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) == 0 && ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) == 0))
    {
        printf ("ERROR! - Wrong parameter settings - (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM) = %d, ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM) = %d\n",
        (_BLOCK_K_SIZE*_BLOCK_M_SIZE)%_THREAD_NUM), ((_BLOCK_K_SIZE*_BLOCK_N_SIZE)%_THREAD_NUM)); 
        exit(0);
    }
    #endif

    dim3 gridDim (M/_BLOCK_M_SIZE + ((M%_BLOCK_M_SIZE) > 0), N/_BLOCK_N_SIZE + ((N%_BLOCK_N_SIZE) > 0), num_heads*batch_size);
    dim3 blockDim ((_BLOCK_M_SIZE / _THREAD_M_SIZE), (_BLOCK_N_SIZE / _THREAD_N_SIZE), 1);
    cuda_v_attention_kernel<<<gridDim, blockDim, 0, stream>>> (num_heads, num_hidden, num_seq,
        M, N, K, input_2, ldv, input_1, ldb, output, ldc);
}
void cuda_residual (const float *input_1, const float *input_2, float *output, unsigned int num_elements
    , LAYER_ACT activation_type, hipStream_t stream)
{
    dim3 gridDim (num_elements/_BLOCK_RESIDUAL_SIZE + ((num_elements%_BLOCK_RESIDUAL_SIZE) > 0), 1, 1);
    dim3 blockDim (_BLOCK_RESIDUAL_SIZE, 1, 1);
    cuda_residual_kernel<<<gridDim, blockDim, 0, stream>>> (num_elements, input_1, input_2, output, activation_type);
}
void cuda_layernorm (const float *input, const float *weight, const float *bias, 
    float *output, unsigned int N, unsigned int M, unsigned int ldb, unsigned int ldc, hipStream_t stream)
{
    dim3 prob_gridDim (N/_BLOCK_LAYERNORM_SIZE + ((N%_BLOCK_LAYERNORM_SIZE) > 0), 1, 1);
    dim3 prob_blockDim (_BLOCK_LAYERNORM_SIZE, 1, 1);
    cuda_layernorm_kernel<<<prob_gridDim, prob_blockDim, 0, stream>>> (input, weight, bias, output, N, M, ldb, ldc);
}
}